#include <hip/hip_runtime.h>
// function just adds vectors A and B and stores result in C
// [0, 1, 2, 3] + [4, 5, 6, 7] = [4, 6, 8, 10]
void vecAdd(float* A, float* B, float* C, int n) {
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    // Allocate device memory for A, B, and C
    // Copy A and B to device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Part 2: Call Kernel - to launch a grid of threads on GPU
    // perform actual vector addition


    // Part 3: Copy C from the device memory
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}
int main() {

}